#include "test/IntervalTest.cuh"
#include "test/gemmGPUCTest.cuh"
#include "test/gemmGPUPyTest.cuh"
#include "test/mmaGPUCTest.cuh"
#include "src/mma.cuh"



int main() {
    /// Interval test
    printf("--------Testing Interval--------\n");
    hasSaferMultiplication<<<1, 1>>>();
    hipDeviceSynchronize();
    hasSaferAddition<<<1,1>>>();
    hipDeviceSynchronize();

    /// gemmGPUCTest
    printf("--------Testing gemmGPUCUsingGPUPtr API--------\n");
    GemmCalculatesCorrectly<float>();
    GemmCalculatesCorrectly<double>();
    GemmCalculatesCorrectly<Interval<float>>();
    // do not why, but this does not work:
    GemmCalculatesCorrectly<Interval<double>>();

    /// gemmGPUPyTest
    printf("--------Testing gemmGPUPy API--------\n");
    canCallGemmGPUC<float>();
    canCallGemmGPUC<double>();
    canCallGemmGPUC<Interval<float>>();
    canCallGemmGPUC<Interval<double>>();

    /// mmaGPUCTest
    printf("--------Testing gemmGPUCUsingGPUPtr API--------\n");
    mmaGPUCalculatesCorrectly<float>();
    mmaGPUCalculatesCorrectly<double>();
    mmaGPUCalculatesCorrectly<Interval<float>>();
    // do not why, but this does not work:
    mmaGPUCalculatesCorrectly<Interval<double>>();
    return 0;
}